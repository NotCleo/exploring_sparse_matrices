#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define ROWS 125000     // Matrix rows (125K)
#define COLS 125000     // Matrix columns (125K)
#define SPARSITY 0.00005 // Adjusted to ~15,625 non-zeros
#define MAX_NNZ 31250LL // 15,625 * 2 as long long
#define THREADS_PER_BLOCK 256

void generate_sparse_matrix(double *val, int *col_idx, int *row_ptr, int *nnz) {
    *nnz = 0;
    row_ptr[0] = 0;
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++) {
            if ((rand() % 1000000) / 1000000.0 < SPARSITY) {
                if (*nnz >= MAX_NNZ) {
                    printf("Warning: Exceeded MAX_NNZ, truncating at %lld\n", (long long)MAX_NNZ);
                    row_ptr[i + 1] = *nnz;
                    for (int k = i + 1; k <= ROWS; k++) row_ptr[k] = *nnz;
                    return;
                }
                val[*nnz] = rand() % 10 + 1;
                col_idx[*nnz] = j;
                (*nnz)++;
            }
        }
        row_ptr[i + 1] = *nnz;
    }
}

__global__ void spmv_kernel(const double *val, const int *col_idx, const int *row_ptr,
                            const double *x, double *y, int rows) {
    extern __shared__ double shared_x[];
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadIdx.x; i < COLS; i += blockDim.x) {
        if (i < COLS) shared_x[i] = x[i];
    }
    __syncthreads();

    if (row < rows) {
        double sum = 0.0;
        for (int j = row_ptr[row]; j < row_ptr[row + 1]; j++) {
            sum += val[j] * shared_x[col_idx[j]];
        }
        y[row] = sum;
    }
}

int main() {
    int i, j, nnz;
    float kernel_time, total_time;

    double *h_val = (double*)malloc(MAX_NNZ * sizeof(double));
    int *h_col_idx = (int*)malloc(MAX_NNZ * sizeof(int));
    int *h_row_ptr = (int*)malloc((ROWS + 1) * sizeof(int));
    double *h_x = (double*)malloc(COLS * sizeof(double));
    double *h_y = (double*)malloc(ROWS * sizeof(double));

    if (!h_val || !h_col_idx || !h_row_ptr || !h_x || !h_y) {
        printf("Memory allocation failed\n");
        exit(1);
    }

    srand(42);
    for (i = 0; i < COLS; i++) h_x[i] = rand() % 10;
    for (i = 0; i < ROWS; i++) h_y[i] = 0;
    generate_sparse_matrix(h_val, h_col_idx, h_row_ptr, &nnz);

    for (i = 0; i < ROWS; i++) {
        for (j = h_row_ptr[i]; j < h_row_ptr[i + 1]; j++) {
            volatile double temp = h_val[j] * h_x[h_col_idx[j]];
            (void)temp;
        }
    }

    double start_seq = omp_get_wtime();
    for (i = 0; i < ROWS; i++) {
        h_y[i] = 0;
        for (j = h_row_ptr[i]; j < h_row_ptr[i + 1]; j++) {
            h_y[i] += h_val[j] * h_x[h_col_idx[j]];
        }
    }
    double end_seq = omp_get_wtime();
    double seq_time = end_seq - start_seq;

    double *d_val, *d_x, *d_y;
    int *d_col_idx, *d_row_ptr;
    hipMalloc(&d_val, MAX_NNZ * sizeof(double));
    hipMalloc(&d_col_idx, MAX_NNZ * sizeof(int));
    hipMalloc(&d_row_ptr, (ROWS + 1) * sizeof(int));
    hipMalloc(&d_x, COLS * sizeof(double));
    hipMalloc(&d_y, ROWS * sizeof(double));

    if (hipGetLastError() != hipSuccess) {
        printf("CUDA memory allocation failed\n");
        exit(1);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipMemcpy(d_val, h_val, MAX_NNZ * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, h_col_idx, MAX_NNZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, h_row_ptr, (ROWS + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, COLS * sizeof(double), hipMemcpyHostToDevice);

    int grid_size = (ROWS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipEvent_t kernel_start, kernel_stop;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_stop);
    hipEventRecord(kernel_start);
    spmv_kernel<<<grid_size, THREADS_PER_BLOCK, COLS * sizeof(double)>>>(d_val, d_col_idx, d_row_ptr, d_x, d_y, ROWS);
    hipEventRecord(kernel_stop);
    hipEventSynchronize(kernel_stop);
    hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);

    hipMemcpy(h_y, d_y, ROWS * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&total_time, start, stop);

    printf("Sequential CPU Time: %.6f seconds\n", seq_time);
    printf("CUDA Kernel Time: %.6f seconds\n", kernel_time / 1000.0);
    printf("Total CUDA Time (including transfers): %.6f seconds\n", total_time / 1000.0);
    printf("Speedup (Kernel vs Sequential): %.2fx\n", seq_time / (kernel_time / 1000.0));
    printf("Speedup (Total vs Sequential): %.2fx\n", seq_time / (total_time / 1000.0));
    printf("Non-zero Elements: %d (%.4f%% sparsity)\n", nnz, (float)nnz / ((long long)ROWS * COLS) * 100);
    printf("Grid Size: %d blocks, Block Size: %d threads\n", grid_size, THREADS_PER_BLOCK);

    hipFree(d_val); hipFree(d_col_idx); hipFree(d_row_ptr);
    hipFree(d_x); hipFree(d_y);
    free(h_val); free(h_col_idx); free(h_row_ptr);
    free(h_x); free(h_y);
    hipEventDestroy(start); hipEventDestroy(stop);
    hipEventDestroy(kernel_start); hipEventDestroy(kernel_stop);

    return 0;
}
