#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ROWS 1000    // Matrix rows
#define COLS 1000    // Matrix columns
#define SPARSITY 0.01 // 1% non-zero elements
#define THREADS_PER_BLOCK 256

// Generate a sparse matrix in CSR format (same as before)
void generate_sparse_matrix(double *val, int *col_idx, int *row_ptr, int *nnz) {
    *nnz = 0;
    row_ptr[0] = 0;
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++) {
            if ((rand() % 10000) / 10000.0 < SPARSITY) {
                val[*nnz] = rand() % 10 + 1;
                col_idx[*nnz] = j;
                (*nnz)++;
            }
        }
        row_ptr[i + 1] = *nnz;
    }
}

// CUDA kernel for SpMV
__global__ void spmv_kernel(const double *val, const int *col_idx, const int *row_ptr, 
                            const double *x, double *y, int rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        double sum = 0.0;
        for (int j = row_ptr[row]; j < row_ptr[row + 1]; j++) {
            sum += val[j] * x[col_idx[j]];
        }
        y[row] = sum;
    }
}

int main() {
    int i, nnz;
    float kernel_time, total_time;

    // Host arrays
    double *h_val = (double*)malloc(ROWS * COLS * sizeof(double));
    int *h_col_idx = (int*)malloc(ROWS * COLS * sizeof(int));
    int *h_row_ptr = (int*)malloc((ROWS + 1) * sizeof(int));
    double *h_x = (double*)malloc(COLS * sizeof(double));
    double *h_y = (double*)malloc(ROWS * sizeof(double));

    // Initialize data
    for (i = 0; i < COLS; i++) h_x[i] = rand() % 10;
    for (i = 0; i < ROWS; i++) h_y[i] = 0;
    generate_sparse_matrix(h_val, h_col_idx, h_row_ptr, &nnz);

    // Device arrays
    double *d_val, *d_x, *d_y;
    int *d_col_idx, *d_row_ptr;
    hipMalloc(&d_val, ROWS * COLS * sizeof(double));
    hipMalloc(&d_col_idx, ROWS * COLS * sizeof(int));
    hipMalloc(&d_row_ptr, (ROWS + 1) * sizeof(int));
    hipMalloc(&d_x, COLS * sizeof(double));
    hipMalloc(&d_y, ROWS * sizeof(double));

    // Copy data to device
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipMemcpy(d_val, h_val, ROWS * COLS * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, h_col_idx, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, h_row_ptr, (ROWS + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, COLS * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 block(THREADS_PER_BLOCK);
    dim3 grid((ROWS + block.x - 1) / block.x);
    hipEvent_t kernel_start, kernel_stop;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_stop);
    hipEventRecord(kernel_start);
    spmv_kernel<<<grid, block>>>(d_val, d_col_idx, d_row_ptr, d_x, d_y, ROWS);
    hipEventRecord(kernel_stop);
    hipEventSynchronize(kernel_stop);
    hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);

    // Copy result back
    hipMemcpy(h_y, d_y, ROWS * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&total_time, start, stop);

    // Output metrics
    printf("CUDA Kernel Time: %f seconds\n", kernel_time / 1000.0);
    printf("Total Time (including transfers): %f seconds\n", total_time / 1000.0);
    printf("Non-zero Elements: %d (%.2f%% sparsity)\n", nnz, (float)nnz / (ROWS * COLS) * 100);
    printf("Grid Size: %d blocks, Block Size: %d threads\n", grid.x, block.x);

    // Sequential CPU for speedup comparison
    double start_seq = omp_get_wtime();
    for (i = 0; i < ROWS; i++) {
        h_y[i] = 0;
        for (int j = h_row_ptr[i]; j < h_row_ptr[i + 1]; j++) {
            h_y[i] += h_val[j] * h_x[h_col_idx[j]];
        }
    }
    double end_seq = omp_get_wtime();
    double seq_time = end_seq - start_seq;
    printf("Sequential CPU Time: %f seconds\n", seq_time);
    printf("Speedup (Total vs Sequential): %.2fx\n", seq_time / (total_time / 1000.0));

    // Cleanup
    hipFree(d_val); hipFree(d_col_idx); hipFree(d_row_ptr);
    hipFree(d_x); hipFree(d_y);
    free(h_val); free(h_col_idx); free(h_row_ptr);
    free(h_x); free(h_y);
    hipEventDestroy(start); hipEventDestroy(stop);
    hipEventDestroy(kernel_start); hipEventDestroy(kernel_stop);

    return 0;
} 
